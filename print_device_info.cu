/*
* The MIT License (MIT)
*
* Copyright (c) 2013 cuda.geek (cuda.geek@gmail.com)
*
* Permission is hereby granted, free of charge, to any person obtaining a copy of
* this software and associated documentation files (the "Software"), to deal in
* the Software without restriction, including without limitation the rights to
* use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
* the Software, and to permit persons to whom the Software is furnished to do so,
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
* FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
* COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER
* IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN
* CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// simple function that print information about device which is currently benchmarked.
// Inspired by deviceQuery CUDA SDK example. see http://docs.nvidia.com/cuda/cuda-samples/index.html#cudalibraries

#include "cudassert.cuh"

void printCudaDeviceInfo(int dev)
{
    int driverVersion = 0, runtimeVersion = 0;

    cuda_assert( hipDriverGetVersion(&driverVersion) );
    cuda_assert( hipRuntimeGetVersion(&runtimeVersion) );

    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
        driverVersion / 1000, driverVersion % 100, runtimeVersion / 1000, runtimeVersion % 100);

    const char *computeMode[] = {
        "Default (multiple host threads can use ::hipSetDevice() with device simultaneously)",
        "Exclusive (only one host thread in one process is able to use ::hipSetDevice() with this device)",
        "Prohibited (no host thread can use ::hipSetDevice() with this device)",
        "Exclusive Process (many threads in one process is able to use ::hipSetDevice() with this device)",
        "Unknown",
        0
    };

    {
        hipDeviceProp_t prop;
        cuda_assert( hipGetDeviceProperties(&prop, dev) );

        printf("\nDevice %d: \"%s\"\n", dev, prop.name);
        printf("  CUDA Capability Major/Minor version number:    %d.%d\n", prop.major, prop.minor);
        printf("  Warp size:                                     %d\n", prop.warpSize);
        printf("  GPU Clock Speed:                               %.2f GHz\n", prop.clockRate * 1e-6f);
        printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
            (float)prop.totalGlobalMem/1048576.0f, (unsigned long long) prop.totalGlobalMem);
        printf("  Total amount of constant memory:               %u bytes\n", (int)prop.totalConstMem);
        printf("  Total amount of shared memory per block:       %u bytes\n", (int)prop.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n", prop.regsPerBlock);
        printf("  Maximum number of threads per block:           %d\n", prop.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n", prop.maxThreadsDim[0],
            prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n", prop.maxGridSize[0],
            prop.maxGridSize[1],  prop.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %u bytes\n", (int)prop.memPitch);
        printf("  Texture alignment:                             %u bytes\n", (int)prop.textureAlignment);
        printf("  Concurrent copy and execution:                 %s with %d copy engine(s)\n",
            (prop.deviceOverlap ? "Yes" : "No"), prop.asyncEngineCount);
        printf("  Run time limit on kernels:                     %s\n", prop.kernelExecTimeoutEnabled ? "Yes" : "No");
        printf("  Integrated GPU sharing Host Memory:            %s\n", prop.integrated ? "Yes" : "No");
        printf("  Support host page-locked memory mapping:       %s\n", prop.canMapHostMemory ? "Yes" : "No");

        printf("  Concurrent kernel execution:                   %s\n", prop.concurrentKernels ? "Yes" : "No");
        printf("  Alignment requirement for Surfaces:            %s\n", prop.surfaceAlignment ? "Yes" : "No");
        printf("  Device has ECC support enabled:                %s\n", prop.ECCEnabled ? "Yes" : "No");
        printf("  Device is using TCC driver mode:               %s\n", prop.tccDriver ? "Yes" : "No");
        printf("  Device supports Unified Addressing (UVA):      %s\n", prop.unifiedAddressing ? "Yes" : "No");
        printf("  Device PCI Bus ID / PCI location ID:           %d / %d\n", prop.pciBusID, prop.pciDeviceID );
        printf("  Compute Mode:\n");
        printf("      %s \n", computeMode[prop.computeMode]);
    }
    fflush(stdout);
}
