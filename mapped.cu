#include "hip/hip_runtime.h"
/*
* The MIT License (MIT)
*
* Copyright (c) 2013-2014 cuda.geek (cuda.geek@gmail.com)
*
* Permission is hereby granted, free of charge, to any  person obtaining a copy of
* this software  and associated  documentation  files (the "Software"), to deal in
* the Software without  restriction, including  without  limitation  the rights to
* use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of
* the Software,  and to permit persons to  whom the Software is furnished to do so,
* subject to the following conditions:
*
* The above  copyright notice  and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE  SOFTWARE IS  PROVIDED "AS IS",  WITHOUT  WARRANTY OF  ANY  KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS
* FOR A PARTICULAR  PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE AUTHORS OR
* COPYRIGHT HOLDERS  BE LIABLE FOR ANY CLAIM,  DAMAGES OR OTHER LIABILITY, WHETHER
* IN  AN  ACTION  OF CONTRACT,  TORT OR OTHERWISE,  ARISING  FROM,  OUT  OF  OR IN
* CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include "cudassert.cuh"
#include "cumib.cuh"

#include <cmath>

using namespace cumib;

__global__ void inc_kernel(unsigned int *cnt)
{
    printf("w %d %u\n", threadIdx.x, *cnt);
    atomicAdd(cnt, 1);
    printf("b %d %u\n", threadIdx.x, *cnt);
}

template<typename T>
__global__ void copy1d1d(const T *src, T *dst)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 300
    const int index = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
#else
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
#endif
    dst[index] = src[index];
}

template<typename T>
__global__ void fillWithTid(T *dst)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    dst[index] = index;
}

enum
{
    GMEM_DEVICE = 1,
    GMEM_HOST   = 2
};

template<int TYPE> struct Dispatcher;
template<> struct Dispatcher<GMEM_HOST> {};
template<> struct Dispatcher<GMEM_DEVICE> {};

template<typename T, typename D> struct MPtr
{
    MPtr(const size_t size, bool fill = false)
    {
        cuda_assert(hipMalloc((void**) &_ptr, sizeof(T) * size));
        if (0 && fill)
        {
            fillWithTid<<<size / CTA_SIZE, CTA_SIZE>>>(_ptr);
            cuda_assert(hipDeviceSynchronize());
            cuda_assert(hipGetLastError());
        }
    }

    T* ptr() const {return _ptr;}

    const char* space() {return "gpu";}

    ~MPtr()
    {
        cuda_assert(hipFree((void*) _ptr));
    }

private:
    T* _ptr;

    enum name {CTA_SIZE = 128};
};

template<typename T> struct MPtr<T, Dispatcher<GMEM_HOST> >
{
    MPtr(const size_t size, bool fill = false)
    {
        // hipHostMallocMapped
        cuda_assert(hipHostAlloc((void**) &_ptr, sizeof(T) * size, hipHostMallocDefault));
        if (fill)
        {
            for (size_t i = 0; i < size; ++i)
                _ptr[i] = i;
        }
    }

    T* ptr() const {return _ptr;}

    const char* space() {return "host";}

    ~MPtr()
    {
        cuda_assert(hipHostFree((void*) _ptr));
    }

private:
    T* _ptr;
};

static int getMajorCC(int deviceId)
{
    hipDeviceProp_t prop;
    cuda_assert( hipGetDeviceProperties(&prop, deviceId) );
    // in bytes
    return prop.major;
}

static dim3 declareGrid(int size, int cta_size)
{
    int deviceId = 0;
    cuda_assert(hipGetDevice(&deviceId));
    const int major_cc = getMajorCC(deviceId);

    dim3 grid(size/cta_size);
    if((major_cc < 3) && (size/cta_size) >= 65536)
        grid = dim3( static_cast<int>(sqrt(size/cta_size)), static_cast<int>(sqrt(size/cta_size)));

    return grid;
}

template<typename Pi, typename Po>
static void run_mapped_test(const size_t min_array_size, const size_t  max_array_size)
{
    Pi src(max_array_size, true);
    Po dst(max_array_size);

    HostTimer timer;

    static const int cta_size = 128;

    for (size_t size = min_array_size; size < max_array_size; size *=2)
    {
        printf("run: %zi\t", size);

        dim3 grid = declareGrid(size, cta_size);

        timer.go();
        copy1d1d<<<grid, cta_size>>>(src.ptr(), dst.ptr());
        cuda_assert(hipDeviceSynchronize());
        cuda_assert(hipGetLastError());

        float mtime = timer.measure();
        printf("%s -> %s:\t %f ms\n", src.space(), dst.space(), mtime);
    }
}

template<typename T>
static void run_mapped_tests(const size_t min_array_size, const size_t  max_array_size)
{
    run_mapped_test<MPtr<T,Dispatcher<GMEM_DEVICE> >, MPtr<T,Dispatcher<GMEM_DEVICE> > >(min_array_size, max_array_size);
    run_mapped_test<MPtr<T,Dispatcher<GMEM_DEVICE> >, MPtr<T,Dispatcher<GMEM_HOST> > >(min_array_size, max_array_size);
    run_mapped_test<MPtr<T,Dispatcher<GMEM_HOST> >,   MPtr<T,Dispatcher<GMEM_DEVICE> > >(min_array_size, max_array_size);
    run_mapped_test<MPtr<T,Dispatcher<GMEM_HOST> >,   MPtr<T,Dispatcher<GMEM_HOST> > >(min_array_size, max_array_size);
}

int main(int argc, char **argv)
{
    typedef unsigned int test_t;

    int deviceId = 0;

    if (argc >= 2)
        deviceId = atoi(argv[1]);

    cuda_assert(hipSetDevice(deviceId));
    printCudaDeviceInfo(deviceId);

    static const size_t min_array_size = static_cast<size_t>(std::pow(2., 15.));
    static const size_t max_array_size = static_cast<size_t>(std::pow(2., 27));
    run_mapped_tests<test_t>(min_array_size, max_array_size);
}